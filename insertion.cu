// heavy assistance provided from nVidia's CUDA documentation and `vectorAdd.cu` piece of sample code
#include <stdio.h>
#include <sys/time.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

int* generate_array(int); // prototypes at the top of a non-header, because I hate C.
char* run_insertion_sort(int); // wraps the cuda_insertion_sort function

// this is quite possibly the stupidest piece of code I've written
// this is a single CUDA block for doing insertion sort
// insertion sort is not a parallelizable algorithm.
__global__ void cuda_insertion_sort(int *array, int num_elements) {
	int temp;
    for (int i = 1; i < num_elements; i++) {
      for(int j = i ; j > 0 ; j--){
        if(array[j] < array[j-1]){
          temp = array[j];
          array[j] = array[j-1];
          array[j-1] = temp;
        }
      }
    }
}

int main(void) {
    FILE *f;
    f = fopen("cuda_insertion.txt", "w");
	for(int i = 1000; i < 11000; i+= 1000) {
        printf("%d ", i);
        fprintf(f, "%d ", i);
        char* return_time = run_insertion_sort(i);
        fprintf(f, "%s ", return_time);
        printf("%s ", return_time);
        fflush(stdout);
        free(return_time);
        printf("\n");
        fprintf(f, "\n");
	}
    hipError_t err = hipDeviceReset();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    fflush(f);
    fclose(f);
    printf("Done\n");
    return 0;
}

char* run_insertion_sort(int num_elements) {
	// initialize host's elements
    hipError_t err = hipSuccess;
    int* host_array = generate_array(num_elements);

    // initialize CUDA device's element
    int* cuda_array = NULL;
    size_t size = num_elements * sizeof(int);
    err = hipMalloc((void **)&cuda_array, size);
    if (err != hipSuccess) {  // check for errors on memory allocation
        fprintf(stderr, "Failed to allocate memory for array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	struct timeval tval_before, tval_after, tval_result; // declare some timing info
	gettimeofday(&tval_before, NULL);

    // copy the host element onto the CUDA device's element
    err = hipMemcpy(cuda_array, host_array, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {  // check for errors on memory copy over to device
        fprintf(stderr, "Failed to copy array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    cuda_insertion_sort<<<1,1>>>(cuda_array, num_elements);  // execute the kernel

    err = hipGetLastError();  // check for any errors during kernel execution
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch `cuda_insertion_sort` kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // copy the result back from the CUDA device
    err = hipMemcpy(host_array, cuda_array, size, hipMemcpyDeviceToHost);  // this is a synchronous function.
    gettimeofday(&tval_after, NULL);
	timersub(&tval_after, &tval_before, &tval_result);  // finish up the timing
    if (err != hipSuccess) {  // check for any errors on memory copy back to host
        fprintf(stderr, "Failed to copy array from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // and clean up
	err = hipFree(cuda_array);
    if (err != hipSuccess) {  // check for any errors on freeing the memory
        fprintf(stderr, "Failed to free device array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    free(host_array);

    // return info on the time spent
    char* return_string = (char*)malloc(100 * sizeof(char));
    sprintf(return_string, "%ld%03ld", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec / 1000);
    return return_string;
}

int* generate_array(int array_length) {
	int *return_var = (int*)malloc(sizeof(int) * array_length);

    for (int i = array_length - 1; i >= 0; i--) {
      return_var[array_length - i - 1] = i;
    }
    return return_var;
}


